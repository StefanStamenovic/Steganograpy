#include "hip/hip_runtime.h"
	#include <stdio.h>
#include <string>
#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "fwhtKernel.cuh"

#define _val(ptr) *ptr
//----------------------------------------------------------------------------------------
// EXCEPTION HANDLING
//----------------------------------------------------------------------------------------

class Exception {
public:
	static char* message;
public:
	Exception() {
		if (this->message != NULL)
			delete this->message;
		this->message = NULL;
		message = new char[1];
		strcpy(this->message, "");
	}
	Exception(const char* message) {
		if (this->message != NULL)
			delete this->message;
		this->message = NULL;
		this->message = new char[strlen(message) + 1];
		strcpy(this->message, message);
	}
	Exception(const char* message, hipError_t error) {
		if (this->message != NULL)
			delete this->message;
		const char* cudaErrorMessage = hipGetErrorString(error);
		this->message = NULL;
		this->message = new char[strlen(message) + strlen(cudaErrorMessage) + 1];
		strcpy(this->message, message);
		strcpy(&(this->message[strlen(message)]), cudaErrorMessage);
	}
};
char* Exception::message = NULL;

///////////////////////////////////////////////////////////////////
// Returns last exception message.
///////////////////////////////////////////////////////////////////
extern "C" void _declspec(dllexport) _stdcall GetLastException(char* _exception)
{
	strcpy(_exception, Exception::message);
	Exception no_exception("There is no CUDA exception.");
}

//----------------------------------------------------------------------------------------
// DEVICE INFO
//----------------------------------------------------------------------------------------

///////////////////////////////////////////////////////////////////
// Checks for is CUDA available.
///////////////////////////////////////////////////////////////////
extern "C" bool _declspec(dllexport) _stdcall CUDA_Check() {
	int deviceCount = 0;
	hipError_t cuda_error = hipGetDeviceCount(&deviceCount);
	if (cuda_error != hipSuccess || deviceCount < 1)
		return false;
	return true;
}

///////////////////////////////////////////////////////////////////
// Compute num of cores from version.
///////////////////////////////////////////////////////////////////
inline int _ConvertSMVer2Cores(
	int major,
	int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
		{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
		{ 0x32, 192 }, // Kepler Generation (SM 3.2) GK10x class
		{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
		{ 0x37, 192 }, // Kepler Generation (SM 3.7) GK21x class
		{ 0x50, 128 }, // Maxwell Generation (SM 5.0) GM10x class
		{ 0x52, 128 }, // Maxwell Generation (SM 5.2) GM20x class
		{ 0x53, 128 }, // Maxwell Generation (SM 5.3) GM20x class
		{ 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
		{ 0x61, 128 }, // Pascal Generation (SM 6.1) GP10x class
		{ 0x62, 128 }, // Pascal Generation (SM 6.2) GP10x class
		{ 0x70, 64 }, // Volta Generation (SM 7.0) GV100 class

		{ -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// Can't compute cores number 
	return -1;
}

///////////////////////////////////////////////////////////////////
// Getting info about best cuda device.
///////////////////////////////////////////////////////////////////
extern "C" void _declspec(dllexport) _stdcall CUDA_BestDeviceInfo(
	int* device,
	char* name,
	float* driver_version,
	float* runtime_version,
	float* capability_version,
	int* global_memory,
	int* memory_clock_rate,
	int* multyprocessors_num,
	int* memory_bus_width,
	int* cuda_cores_num,
	int* clock_rate
)throw(char*) {
	// Getting device count
	int deviceCount = 0;
	hipError_t cuda_error = hipGetDeviceCount(&deviceCount);
	if (cuda_error != hipSuccess  || deviceCount < 1)
		throw Exception("There is no cuda enabled devices.");
	
	// Finding best CUDA enabled device
	hipDeviceProp_t bestDeviceProp;
	int bestDeviceIndx = 0;
	cuda_error = hipGetDeviceProperties(&bestDeviceProp, bestDeviceIndx);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to get device properties.");
	for (int deviceIndx = 1; deviceIndx < deviceCount; deviceIndx++)
	{
		hipDeviceProp_t deviceProp;
		cuda_error = hipGetDeviceProperties(&deviceProp, 0);
		if (cuda_error != hipSuccess)
			throw Exception("Failed to get device properties.");

		if (bestDeviceProp.major < deviceProp.major)
			bestDeviceIndx = deviceIndx;
		else if (bestDeviceProp.major == deviceProp.major)
			if(bestDeviceProp.clockRate < deviceProp.clockRate)
				bestDeviceIndx = deviceIndx;
	}
	cuda_error = hipGetDeviceProperties(&bestDeviceProp, bestDeviceIndx);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to get device properties.");

	cuda_error = hipSetDevice(bestDeviceIndx);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to set CUDA device.");

	// Gettin device driver version
	int driverVersion = 0;
	cuda_error = hipDriverGetVersion(&driverVersion);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to get device driver version.");
	_val(driver_version) = driverVersion / 1000 + (driverVersion % 100) / 100.0f;
	// Gettin device driver version
	int runtimeVersion = 0;
	cuda_error = hipRuntimeGetVersion(&runtimeVersion);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to get device runtime version.");
	_val(runtime_version) = driverVersion / 1000.0f + (driverVersion % 100) / 100.0f;

	// Gettin device properties
	try {
		_val(device) = bestDeviceIndx;
		strcpy(name, bestDeviceProp.name);
		_val(capability_version) = (float)bestDeviceProp.major + (bestDeviceProp.minor) / 10.f;
		_val(global_memory) = bestDeviceProp.totalGlobalMem / 1048576.0f;
		_val(memory_clock_rate) = bestDeviceProp.memoryClockRate;
		_val(multyprocessors_num) = bestDeviceProp.multiProcessorCount;
		_val(memory_bus_width) = bestDeviceProp.memoryBusWidth;
		_val(cuda_cores_num) = _ConvertSMVer2Cores(bestDeviceProp.major, bestDeviceProp.minor) * bestDeviceProp.multiProcessorCount;
		_val(clock_rate) = bestDeviceProp.clockRate;
	}
	catch (...)
	{
		throw Exception("Failed to get CUDA device properties.");
	}
}


extern "C" int _declspec(dllexport) _stdcall CUDA_DeviceNum()
{
	return 10;
}



///////////////////////////////////////////////////////////////////
// Autocorrelation
///////////////////////////////////////////////////////////////////


extern "C" void _declspec(dllexport) _stdcall CUDA_PackBytes(unsigned char* imageBytes, int imageBytesLength, unsigned char* fileBytes, int fileBytesLength)
throw (char*)
{
	hipError_t err = hipSuccess;

	size_t sizeIn = fileBytesLength * sizeof(unsigned char);
	size_t sizeOut = imageBytesLength * sizeof(unsigned char);

	unsigned char* d_inData;
	err = hipMalloc((void**)&d_inData, sizeIn);
	if (err != hipSuccess)
	{
		throw Exception("Failed to allocate device vector - error code: ", err);
	}

	err = hipMemcpy(d_inData, fileBytes, sizeIn, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from host to device - error code: ", err);
	}

	unsigned char* d_outData;
	err = hipMalloc((void**)&d_outData, sizeOut);
	if (err != hipSuccess)
	{
		throw Exception("Failed to allocate device vector - error code: ", err);
	}

	err = hipMemcpy(d_outData, imageBytes, sizeOut, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from host to device - error code: ", err);
	}

	int threadsPerBlock = 512;
	int blocksPerGrid = (int)((fileBytesLength * 8 + threadsPerBlock - 1) / threadsPerBlock);
	dim3 threadsPerBlockDim(64, 8);
	packBits << <blocksPerGrid, threadsPerBlockDim >> >(d_inData, fileBytesLength, d_outData, imageBytesLength);

	hipMemcpy(imageBytes, d_outData, sizeOut, hipMemcpyDeviceToHost);

	hipFree(d_inData);
	hipFree(d_outData);
}

extern "C" void _declspec(dllexport) _stdcall CUDA_UnpackBytes(unsigned char* imageBytes, int imageBytesLength, unsigned char* fileBytes, int fileBytesLength)
throw (char*)
{
	hipError_t err = hipSuccess;

	size_t sizeOut = fileBytesLength * sizeof(unsigned char);
	size_t sizeIn = imageBytesLength * sizeof(unsigned char);

	unsigned char* d_inData;
	err = hipMalloc((void**)&d_inData, sizeIn);
	if (err != hipSuccess)
	{
		throw Exception("Failed to allocate device vector - error code: ", err);
	}

	err = hipMemcpy(d_inData, imageBytes, sizeIn, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from host to device - error code: ", err);
	}

	unsigned char* d_outData;
	err = hipMalloc((void**)&d_outData, sizeOut);
	if (err != hipSuccess)
	{
		throw Exception("Failed to allocate device vector - error code: ", err);
	}

	err = hipMemcpy(d_outData, fileBytes, sizeOut, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from host to device - error code: ", err);
	}

	int threadsPerBlock = 512;
	int blocksPerGrid = (int)((fileBytesLength * 8 + threadsPerBlock - 1) / threadsPerBlock);
	dim3 threadsPerBlockDim(64, 8);
	unpackBits << <blocksPerGrid, threadsPerBlockDim >> >(d_inData, imageBytesLength, d_outData, fileBytesLength);

	hipMemcpy(fileBytes, d_outData, sizeOut, hipMemcpyDeviceToHost);

	hipFree(d_inData);
	hipFree(d_outData);
}

void CalculateAutoCorrelation(unsigned char* imageBytes, int imageBytesLength, int** autoCorrelation, int* autoCorrelationLength);

extern "C" void main()
{

}

extern "C" void _declspec(dllexport) _stdcall CUDA_CalculateAutoCorrelation(unsigned char* imageBytesOriginal, int imageBytesOriginalLength, unsigned char* imageBytesPacked, int imageBytesPackedLength, int* returnData)
{
	hipError_t err = hipSuccess;
	
	int* originalAutoCorr;
	int originalAutoCorrLength;

	int * packedAutoCorr;
	int packedAutoCorrLength;

	CalculateAutoCorrelation(imageBytesOriginal, imageBytesOriginalLength, &originalAutoCorr, &originalAutoCorrLength);
	CalculateAutoCorrelation(imageBytesPacked, imageBytesPackedLength, &packedAutoCorr, &packedAutoCorrLength);

	size_t sizeOut = packedAutoCorrLength * sizeof(int);
	size_t sizeIn = originalAutoCorrLength * sizeof(int);

	int* h_retData = (int *)malloc(sizeIn);
	memset(h_retData, 0, sizeIn);

	int* d_inData;
	err = hipMalloc((void**)&d_inData, sizeIn);
	if (err != hipSuccess)
	{
		throw Exception("Failed to allocate device vector - error code: ", err);
	}

	err = hipMemcpy(d_inData, originalAutoCorr, sizeIn, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from host to device - error code: ", err);
	}

	int* d_outData;
	err = hipMalloc((void**)&d_outData, sizeOut);
	if (err != hipSuccess)
	{
		throw Exception("Failed to allocate device vector - error code: ", err);
	}

	err = hipMemcpy(d_outData, packedAutoCorr, sizeOut, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from host to device - error code: ", err);
	}

	int threadsPerBlock = 512;
	int blocksPerGrid = (int)((originalAutoCorrLength + threadsPerBlock - 1) / threadsPerBlock);
	vectorSubstraction << <blocksPerGrid, threadsPerBlock >> >(d_inData, d_outData, originalAutoCorrLength);

	err = hipMemcpy(h_retData, d_inData, sizeIn, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from device to host - error code: ", err);
	}

	hipFree(d_inData);
	hipFree(d_outData);

	memcpy(returnData, h_retData, sizeIn);
	free(h_retData);
}

void CalculateAutoCorrelation(unsigned char* imageBytes, int imageBytesLength, int** autoCorrelation, int* autoCorrelationLength)
{
	hipError_t err = hipSuccess;

	int logLength = (int)ceil(log2(imageBytesLength / 8));
	int outLength = 1 << logLength;
	*autoCorrelationLength = outLength;

	size_t sizeIn = imageBytesLength * sizeof(unsigned char);
	size_t sizeOut = outLength * sizeof(int);

	int* h_outData = (int *)malloc(sizeOut);
	memset(h_outData, 0, sizeOut);

	unsigned char* d_inData;
	err = hipMalloc((void**)&d_inData, sizeIn);
	if (err != hipSuccess)
	{
		throw Exception("Failed to allocate device vector - error code: ", err);
	}

	err = hipMemcpy(d_inData, imageBytes, sizeIn, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from host to device - error code: ", err);
	}

	int* d_outData;
	err = hipMalloc((void**)&d_outData, sizeOut);
	if (err != hipSuccess)
	{
		throw Exception("Failed to allocate device vector - error code: ", err);
	}

	err = hipMemcpy(d_outData, h_outData, sizeOut, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from host to device - error code: ", err);
	}

	int threadsPerBlock = 512;
	int blocksPerGrid = (int)((outLength + threadsPerBlock - 1) / threadsPerBlock);
	extractBits << <blocksPerGrid, threadsPerBlock >> >(d_inData, imageBytesLength, d_outData, outLength);
	
	fwtBatchGPU(d_outData, 1, logLength);
	vectorSquare << <blocksPerGrid, threadsPerBlock >> >(d_outData, outLength);
	fwtBatchGPU(d_outData, 1, logLength);
	vectorMultiplyScalar << <blocksPerGrid, threadsPerBlock >> >(d_outData, outLength, logLength);
	
	err = hipMemcpy(h_outData, d_outData, sizeOut, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		throw Exception("Failed to copy vector from device to host - error code: ", err);
	}

	hipFree(d_inData);
	hipFree(d_outData);

	*autoCorrelation = h_outData;
}