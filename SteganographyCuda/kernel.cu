#include <stdio.h>
#include <string>
#include "hip/hip_runtime.h"

#define _val(ptr) *ptr
//----------------------------------------------------------------------------------------
// EXCEPTION HANDLING
//----------------------------------------------------------------------------------------

class Exception {
public:
	static char* message;
public:
	Exception() {
		if (this->message != NULL)
			delete this->message;
		this->message = NULL;
		message = new char[1];
		strcpy(this->message, "");
	}
	Exception(const char* message) {
		if (this->message != NULL)
			delete this->message;
		this->message = NULL;
		this->message = new char[strlen(message) + 1];
		strcpy(this->message, message);
	}
};
char* Exception::message = NULL;

///////////////////////////////////////////////////////////////////
// Returns last exception message.
///////////////////////////////////////////////////////////////////
extern "C" void _declspec(dllexport) _stdcall GetLastException(char* _exception)
{
	strcpy(_exception, Exception::message);
	Exception no_exception("There is no CUDA exception.");
}

//----------------------------------------------------------------------------------------
// DEVICE INFO
//----------------------------------------------------------------------------------------

///////////////////////////////////////////////////////////////////
// Checks for is CUDA available.
///////////////////////////////////////////////////////////////////
extern "C" bool _declspec(dllexport) _stdcall CUDA_Check() {
	int deviceCount = 0;
	hipError_t cuda_error = hipGetDeviceCount(&deviceCount);
	if (cuda_error != hipSuccess || deviceCount < 1)
		return false;
	return true;
}

///////////////////////////////////////////////////////////////////
// Compute num of cores from version.
///////////////////////////////////////////////////////////////////
inline int _ConvertSMVer2Cores(
	int major,
	int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
		{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
		{ 0x32, 192 }, // Kepler Generation (SM 3.2) GK10x class
		{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
		{ 0x37, 192 }, // Kepler Generation (SM 3.7) GK21x class
		{ 0x50, 128 }, // Maxwell Generation (SM 5.0) GM10x class
		{ 0x52, 128 }, // Maxwell Generation (SM 5.2) GM20x class
		{ 0x53, 128 }, // Maxwell Generation (SM 5.3) GM20x class
		{ 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
		{ 0x61, 128 }, // Pascal Generation (SM 6.1) GP10x class
		{ 0x62, 128 }, // Pascal Generation (SM 6.2) GP10x class
		{ 0x70, 64 }, // Volta Generation (SM 7.0) GV100 class

		{ -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// Can't compute cores number 
	return -1;
}

///////////////////////////////////////////////////////////////////
// Getting info about best cuda device.
///////////////////////////////////////////////////////////////////
extern "C" void _declspec(dllexport) _stdcall CUDA_BestDeviceInfo(
	int* device,
	char* name,
	float* driver_version,
	float* runtime_version,
	float* capability_version,
	int* global_memory,
	int* memory_clock_rate,
	int* multyprocessors_num,
	int* memory_bus_width,
	int* cuda_cores_num,
	int* clock_rate
)throw(char*) {
	// Getting device count
	int deviceCount = 0;
	hipError_t cuda_error = hipGetDeviceCount(&deviceCount);
	if (cuda_error != hipSuccess  || deviceCount < 1)
		throw Exception("There is no cuda enabled devices.");
	
	// Finding best CUDA enabled device
	hipDeviceProp_t bestDeviceProp;
	int bestDeviceIndx = 0;
	cuda_error = hipGetDeviceProperties(&bestDeviceProp, bestDeviceIndx);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to get device properties.");
	for (int deviceIndx = 1; deviceIndx < deviceCount; deviceIndx++)
	{
		hipDeviceProp_t deviceProp;
		cuda_error = hipGetDeviceProperties(&deviceProp, 0);
		if (cuda_error != hipSuccess)
			throw Exception("Failed to get device properties.");

		if (bestDeviceProp.major < deviceProp.major)
			bestDeviceIndx = deviceIndx;
		else if (bestDeviceProp.major == deviceProp.major)
			if(bestDeviceProp.clockRate < deviceProp.clockRate)
				bestDeviceIndx = deviceIndx;
	}
	cuda_error = hipGetDeviceProperties(&bestDeviceProp, bestDeviceIndx);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to get device properties.");

	cuda_error = hipSetDevice(bestDeviceIndx);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to set CUDA device.");

	// Gettin device driver version
	int driverVersion = 0;
	cuda_error = hipDriverGetVersion(&driverVersion);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to get device driver version.");
	_val(driver_version) = driverVersion / 1000 + (driverVersion % 100) / 100.0f;
	// Gettin device driver version
	int runtimeVersion = 0;
	cuda_error = hipRuntimeGetVersion(&runtimeVersion);
	if (cuda_error != hipSuccess)
		throw Exception("Failed to get device runtime version.");
	_val(runtime_version) = driverVersion / 1000.0f + (driverVersion % 100) / 100.0f;

	// Gettin device properties
	try {
		_val(device) = bestDeviceIndx;
		strcpy(name, bestDeviceProp.name);
		_val(capability_version) = (float)bestDeviceProp.major + (bestDeviceProp.minor) / 10.f;
		_val(global_memory) = bestDeviceProp.totalGlobalMem / 1048576.0f;
		_val(memory_clock_rate) = bestDeviceProp.memoryClockRate;
		_val(multyprocessors_num) = bestDeviceProp.multiProcessorCount;
		_val(memory_bus_width) = bestDeviceProp.memoryBusWidth;
		_val(cuda_cores_num) = _ConvertSMVer2Cores(bestDeviceProp.major, bestDeviceProp.minor) * bestDeviceProp.multiProcessorCount;
		_val(clock_rate) = bestDeviceProp.clockRate;
	}
	catch (...)
	{
		throw Exception("Failed to get CUDA device properties.");
	}
}


extern "C" int _declspec(dllexport) _stdcall CUDA_DeviceNum()
{
	return 10;
}